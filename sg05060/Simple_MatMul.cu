#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// A = m x k 
// B = k x n
// C = m x n

#define m 1024
#define n 4096
#define k 2048
#define BLOCK_SIZE 16

__global__ void Simple_MatMul(float *_a, float *_b, float *_c, int _m, int _n, int _k) {
    
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    
    if( row >= _m || col >= _n)
        return;

    float ret = 0;
    for(int i = 0; i < _k; i++)
        ret += __fmul_rn(_a[(_k*row) + i],_b[(i*_n)+ col]);

    _c[(_n * row + col)] = ret;

}

int main(int argc, char** argv) {

    int size_A = m * k;
    int size_B = k * n;
    int size_C = m * n;

    float *a, *b, *c;
    float *_a, *_b, *_c;

    a = new float[size_A]; memset(a,0,size_A);
    b = new float[size_B]; memset(b,0,size_B);
    c = new float[size_C]; memset(c,0,size_C);

    for (int i = 0; i < size_A; i++) {
        a[i] = rand() % 10 + ((rand() % 100) / 100.0);
    }
    for (int i = 0; i < size_B; i++) {
        b[i] = rand() % 10 + ((rand() % 100) / 100.0);
    }

    hipMalloc(&_a, size_A);
    hipMalloc(&_b, size_B);
    hipMalloc(&_c, size_C);

    hipMemcpy(_a, a, size_A, hipMemcpyHostToDevice);
    hipMemcpy(_b, b, size_B, hipMemcpyHostToDevice);

    dim3 gridDim (ceil((float)m / BLOCK_SIZE), ceil((float)n / BLOCK_SIZE));
    dim3 blockDim (BLOCK_SIZE, BLOCK_SIZE);

    Simple_MatMul<<<gridDim, blockDim>>>(_a,_b,_c,m,n,k);
    hipDeviceSynchronize(); 
    hipMemcpy(c, _c, sizeof(float)*size_C, hipMemcpyDeviceToHost);

    bool result = true;
    for (int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            float ret = 0;
            for(int l = 0; l < k; l++)
                ret += a[(k*i)+l] * b[(l*n) + j];
            if(ret != c[i*n + j]) {
                printf("the result is not matched! (%0.2f, %0.2f)\n"
                ,ret, c[i*n + j]);
                result = false;
            }
        }
    }
    if(result)
        printf("kernel works well!\n");
    return 0;
}